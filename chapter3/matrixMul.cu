
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

float randomFloat(float min, float max) {
  float scale = rand() / (float)RAND_MAX;
  return min + scale * (max - min);
}

void randomMatrix(float *A, int width, int height) {
  for (int row = 0; row < height; row++) {
    for (int col = 0; col < width; col++) {
      A[row * width + col] = randomFloat(-1.f, +1.f);
    }
  }
}

__global__ void matrixMulKernel(const float *A, const float *B, float *C,
                                int n) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < n && row < n) {
    float dotProduct = 0;
    for (int k = 0; k < n; k++) {
      dotProduct += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = dotProduct;
  }
}

// Assume square matrixes
void matrixMulGpu(const float *const A, const float *const B, float *C, int n) {
  float *A_d, *B_d, *C_d;

  int size = n * n * sizeof(float);

  gpuErrchk(hipMalloc((void **)&A_d, size));
  gpuErrchk(hipMalloc((void **)&B_d, size));
  gpuErrchk(hipMalloc((void **)&C_d, size));

  gpuErrchk(hipMemcpy(A_d, A, size, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(B_d, B, size, hipMemcpyHostToDevice));

  // (x, y, z)
  dim3 dimGrid(ceil(n / 16.0), ceil(n / 16.0), 1);
  dim3 dimBlock(16, 16, 1);
  matrixMulKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, n);

  gpuErrchk(hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost));

  gpuErrchk(hipFree(A_d));
  gpuErrchk(hipFree(B_d));
  gpuErrchk(hipFree(C_d));
}

// Assume square matrixes
void matrixMulCpu(const float *const A, const float *const B, float *C, int n) {
  for (int row = 0; row < n; row++) {
    for (int col = 0; col < n; col++) {
      float dotProduct = 0.f;
      for (int k = 0; k < n; k++) {
        dotProduct += A[row * n + k] * B[k * n + col];
      }
      C[row * n + col] = dotProduct;
    }
  }
}

// Can't compare floats
//
// void assertSquareMatrix(float *A, float *B, int n) {
//   for (int row = 0; row < n; row++) {
//     for (int col = 0; col < n; col++) {
//       int idx = row * n + col;
//       printf("%.2f == %.2f (%d)\n", A[idx], B[idx], A[idx] == B[idx]);
//       assert(A[idx] == B[idx]);
//     }
//   }
// }

int main(int argc, char *argv[]) {
  int n; // size of the square matrix

  if (argc != 2) {
    printf("Usage: %s <size>\n", argv[0]);
    return 1;
  }

  n = atoi(argv[1]);

  float *A = (float *)malloc(n * n * sizeof(float));
  float *B = (float *)malloc(n * n * sizeof(float));
  float *C = (float *)malloc(n * n * sizeof(float));
  float *D = (float *)malloc(n * n * sizeof(float));

  randomMatrix(A, n, n);
  randomMatrix(B, n, n);

  // matrixMulCpu(A, B, C, n);
  matrixMulGpu(A, B, D, n);

#ifdef DEBUG
  for (int row = 0; row < n; row++) {
    for (int col = 0; col < n; col++) {
      printf("%.2f ", C[row * n + col]);
    }
    printf("\n");
  }
  printf("\n");
#endif

  return EXIT_SUCCESS;
}
