#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x89, 128},
      {0x90, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

int main(int argc, char *argv[]) {
  if (argc != 1) {
    printf("Usage: %s\n", argv[0]);
    return 1;
  }

  int deviceCount = 0;
  gpuErrchk(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev, driverVersion = 0, runtimeVersion = 0;

  for (dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);

    char msg[256];
    snprintf(msg, sizeof(msg),
             "  Total amount of global memory:                 %.0f MBytes "
             "(%llu bytes)\n",
             static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
             (unsigned long long)deviceProp.totalGlobalMem);
    printf("%s", msg);

    printf("  (%03d) Multiprocessors, (%03d) CUDA Cores/MP:    %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
               deviceProp.multiProcessorCount);
    printf(
        "  GPU Max Clock rate:                            %.0f MHz (%0.2f "
        "GHz)\n",
        deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

    // This is supported in CUDA 5.0 (runtime API device properties)
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
           deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize) {
      printf("  L2 Cache Size:                                 %d bytes\n",
             deviceProp.l2CacheSize);
    }

    printf(
        "  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, "
        "%d), 3D=(%d, %d, %d)\n",
        deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
        deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
        deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
    printf(
        "  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
        deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
    printf(
        "  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d "
        "layers\n",
        deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
        deviceProp.maxTexture2DLayered[2]);

    printf("  Total amount of constant memory:               %zu bytes\n",
           deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %zu bytes\n",
           deviceProp.sharedMemPerBlock);
    printf("  Total shared memory per multiprocessor:        %zu bytes\n",
           deviceProp.sharedMemPerMultiprocessor);
    printf("  Total number of registers available per block: %d\n",
           deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
           deviceProp.maxThreadsPerBlock);
    printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %zu bytes\n",
           deviceProp.memPitch);
    printf("  Texture alignment:                             %zu bytes\n",
           deviceProp.textureAlignment);
    printf(
        "  Concurrent copy and kernel execution:          %s with %d copy "
        "engine(s)\n",
        (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
    printf("  Run time limit on kernels:                     %s\n",
           deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
    printf("  Integrated GPU sharing Host Memory:            %s\n",
           deviceProp.integrated ? "Yes" : "No");
    printf("  Support host page-locked memory mapping:       %s\n",
           deviceProp.canMapHostMemory ? "Yes" : "No");
    printf("  Alignment requirement for Surfaces:            %s\n",
           deviceProp.surfaceAlignment ? "Yes" : "No");
    printf("  Device has ECC support:                        %s\n",
           deviceProp.ECCEnabled ? "Enabled" : "Disabled");
    printf("  Device supports Unified Addressing (UVA):      %s\n",
           deviceProp.unifiedAddressing ? "Yes" : "No");
    printf("  Device supports Managed Memory:                %s\n",
           deviceProp.managedMemory ? "Yes" : "No");
    printf("  Device supports Compute Preemption:            %s\n",
           deviceProp.computePreemptionSupported ? "Yes" : "No");
    printf("  Supports Cooperative Kernel Launch:            %s\n",
           deviceProp.cooperativeLaunch ? "Yes" : "No");
    printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n",
           deviceProp.cooperativeMultiDeviceLaunch ? "Yes" : "No");
    printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
           deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);
  }

  return EXIT_SUCCESS;
}
